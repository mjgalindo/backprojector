#include "hip/hip_runtime.h"

#include <chrono>
#include <math.h>
#include <iostream>
#include <fstream>

#include <thread>
#include <chrono>

#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <hip/hip_complex.h>

#ifdef _WIN32
#define MAKE_DLL_EXPORT __declspec(dllexport)
#endif
#include "backproject_cuda.hpp"
#if __linux__
#include "tqdm.h"
#endif

__device__
float distance(const float *p1, const float *p2) 
{
	// Checking if either is a nullptr reduces performance dramatically!
	// if (p1 == nullptr || p2 == nullptr) return 0.0;
    float buff[3];
    for (int i = 0; i < 3; i++)
    {
        buff[i] = p1[i] - p2[i];
    }

    return norm3df(buff[0], buff[1], buff[2]);
}

__forceinline__ __device__
void compute_distance(const float * voxel_position,
					   const float *laser_pos, 
					   const float *camera_pos, 
					   const pointpair * pair,
					   float * distance_out)
{
	// From the laser to the wall
	float laser_wall_distance = distance(laser_pos, pair->laser_point);
	// From the wall to the current voxel
	float laser_point_voxel_distance = distance(pair->laser_point, voxel_position);
	// From the wall back to the camera
	float cam_wall_distance = distance(pair->cam_point, camera_pos);
	// From the object back to the wall
	float voxel_cam_point_distance = distance(voxel_position, pair->cam_point);
	
	*distance_out = laser_wall_distance + laser_point_voxel_distance + voxel_cam_point_distance + cam_wall_distance;
}

__forceinline__ __device__
void advance_block(const uint32_t *voxels_per_side, 
				   uint32_t * kernel_voxels, 
				   uint32_t & block_id,
				   uint32_t * xyz,
				   uint32_t & voxel_id)
{
	// First needs to find the x y z coordinates of the voxel
	block_id = (blockIdx.x * gridDim.y * gridDim.z +
				blockIdx.y * gridDim.z +
				blockIdx.z) * 3;
	for (uint32_t i = 0; i < 3; i++)
		xyz[i] = kernel_voxels[block_id+i]; // , kernel_voxels[block_id+1], kernel_voxels[block_id+2]};
	
	// Set the next voxel to be computed by this blockIdx in the next call.
	// We advance on the Z axis by the dimensions of the kernel, overflowing to the Y axis and 
	// then to the X axis. This would match row-major access to the 3D array.
	__syncthreads();
	if (threadIdx.x == 0)
	{
		uint32_t* next_xyz = &kernel_voxels[block_id];
		next_xyz[2] = xyz[2] + gridDim.z;
		if (next_xyz[2] >= voxels_per_side[2])
		{
			next_xyz[2] = next_xyz[2] % voxels_per_side[2];
			next_xyz[1] = next_xyz[1] + gridDim.y;
			if (next_xyz[1] >= voxels_per_side[1])
			{
				next_xyz[1] = next_xyz[1] % voxels_per_side[1];
				next_xyz[0] = next_xyz[0] + gridDim.x;
			}
		}
	}
	__syncthreads();
	voxel_id = xyz[0] * voxels_per_side[1] * voxels_per_side[2] +
			   xyz[1] * voxels_per_side[2] + 
			   xyz[2];
}


__global__
void cuda_backprojection_impl(float *transient_data,
                              uint32_t *T,
                              uint32_t *num_pairs,
                              pointpair *scanned_pairs,
                              float *camera_pos,
                              float *laser_pos,
                              float *voxel_volume,
                              float *volume_zero_pos,
                              float *voxel_inc,
                              float *t0,
                              float *deltaT,
							  uint32_t *voxels_per_side,
							  uint32_t *kernel_voxels)
{
	uint32_t block_id, voxel_id, xyz[3];
	advance_block(voxels_per_side, kernel_voxels, block_id, xyz, voxel_id);

	// If the block is outside the volume don't do anything
	if ((xyz[0] >= voxels_per_side[0]) | (xyz[1] >= voxels_per_side[1]) | (xyz[2] >= voxels_per_side[2]))
		return;
	
	extern __shared__ double local_array[];
	double& radiance_sum = local_array[threadIdx.x];
	radiance_sum = 0.0;

	{
		float voxel_position[] = {
			volume_zero_pos[0] + voxel_inc[0] * xyz[0] + voxel_inc[3] * xyz[1] + voxel_inc[6] * xyz[2],
			volume_zero_pos[1] + voxel_inc[1] * xyz[0] + voxel_inc[4] * xyz[1] + voxel_inc[7] * xyz[2],
			volume_zero_pos[2] + voxel_inc[2] * xyz[0] + voxel_inc[5] * xyz[1] + voxel_inc[8] * xyz[2]
		};

		for (uint32_t i = 0; i < *num_pairs / blockDim.x; i++)
		{
			uint32_t pair_index = i * blockDim.x + threadIdx.x;
			float total_distance;
			compute_distance(voxel_position, laser_pos, camera_pos, &scanned_pairs[pair_index], &total_distance);
			uint32_t time_index = round((total_distance - *t0) / *deltaT);
			if (time_index < *T)
			{
				uint32_t tdindex = pair_index * *T + time_index;
				radiance_sum += transient_data[tdindex]; // * distance_attenuation;
			}
		}
	}
    __syncthreads();
	if (threadIdx.x == 0)
	{
		// Compute the reduction in a single thread and write it
		for (int i = 1; i < blockDim.x; i++) {
			local_array[0] += local_array[i];
		}
		voxel_volume[voxel_id] = (float) local_array[0];
	}
    __syncthreads();
}

__global__
void rescale_image(const float* input_image,
				   const uint32_t* input_size,
				   float* output_image,
				   const uint32_t* output_size,
				   const uint32_t* bins)
{
	uint32_t work_rows = output_size[0] / gridDim.x;
	uint32_t work_cols = output_size[1] / blockDim.x;

	for (uint32_t y = blockIdx.x * work_rows; y < blockIdx.x * work_rows + work_rows; ++y)
	{
		for (uint32_t x = threadIdx.x * work_cols; x < max(output_size[1], threadIdx.x * work_cols + work_cols); ++x)
		{
			double sum = 0.0;
			uint32_t input_x = x * *bins;
			uint32_t from = min(0u, input_x - *bins / 2);
			uint32_t to = max(input_x + *bins / 2, input_size[1]);
			for (uint32_t time_bin = from; time_bin < to; time_bin++)
			{
				sum += input_image[y*input_size[1]+time_bin];
			}
			output_image[y*output_size[1]+x] = sum;
		}
	}
}

void rescale_image_cpu(const float* input_image,
					   const uint32_t input_size[2],
					   float* output_image,
					   const uint32_t output_size[2],
					   const uint32_t bins)
{
	for (uint32_t y = 0; y < output_size[0]; ++y)
	{
		for (uint32_t x = 0; x < output_size[1]; ++x)
		{
			double sum = 0.0;
			uint32_t from = min(0u, x - bins / 2);
			uint32_t to = max(input_size[1], x + bins / 2);
			for (uint32_t time_bin = from; time_bin < to; time_bin++)
			{
				sum += input_image[y*input_size[1]+time_bin];
			}
			output_image[y*output_size[1]] = sum;
		}
	}
}

void save_array(std::string name, float* buff, uint32_t tot_size)
{
	std::ofstream of(name, std::ios::binary);
	of.write((char*)buff, tot_size*sizeof(float));
	of.close();
}

void call_cuda_backprojection(const float* transient_chunk,
                              uint32_t transient_size, uint32_t T,
                              const std::vector<pointpair> scanned_pairs,
                              const float* camera_position,
                              const float* laser_position,
                              float* voxel_volume,
                              const uint32_t* voxels_per_side,
                              const float* volume_zero_pos,
                              const float* voxel_inc,
                              float t0,
							  float deltaT,
							  bool rescale_to_voxel_size)
{
	thrust::device_vector<float> transient_chunk_gpu(transient_chunk, transient_chunk + transient_size);
	uint32_t num_pairs = scanned_pairs.size();

	if (rescale_to_voxel_size)
	{
		float diagonal = std::sqrt(voxel_inc[0] * voxel_inc[0] +
								   voxel_inc[1] * voxel_inc[1] +
								   voxel_inc[2] * voxel_inc[2]);

		// We'll be reducing the time dimension scaling_factor times
		float scaling_factor = diagonal / deltaT;
		// Scaling to a bigger image is the opposite of what we'd want.
		// In such a case the voxel resolution is already too high
		if (scaling_factor > 1) 
		{
			uint32_t new_T = T / scaling_factor;

			thrust::device_vector<float> new_transient(new_T * num_pairs);
			thrust::device_vector<uint32_t> orig_size({num_pairs, T});
			thrust::device_vector<uint32_t> new_size(num_pairs, new_T);
			thrust::device_vector<uint32_t> bins({(uint32_t) scaling_factor});

			std::cout << "RESCALING " << num_pairs << 'x' << T << " IMAGE to " << num_pairs << 'x' << new_T << "... " << std::flush;
			std::this_thread::sleep_for(std::chrono::milliseconds(30));
			rescale_image<<<8192, 16>>>(
				thrust::raw_pointer_cast(&transient_chunk_gpu[0]),
				thrust::raw_pointer_cast(&orig_size[0]),
				thrust::raw_pointer_cast(&new_transient[0]),
				thrust::raw_pointer_cast(&new_size[0]),
				thrust::raw_pointer_cast(&bins[0])
			);
			std::cout << "DONE!" << std::endl;
			thrust::host_vector<float> chunk_cpy = transient_chunk_gpu;
			save_array("original.chunk", chunk_cpy.data(), transient_size);
			transient_chunk_gpu = std::move(new_transient);
			save_array("comp.chunk", chunk_cpy.data(), num_pairs*new_T);
			deltaT = deltaT * scaling_factor;
			T = new_T;
			std::cout << "New deltaT " << deltaT << std::endl;
		}
	}

	thrust::device_vector<uint32_t> T_gpu(&T, &T + 1);
	thrust::device_vector<uint32_t> num_pairs_gpu(&num_pairs, &num_pairs + 1);
	thrust::device_vector<pointpair> scanned_pairs_gpu(scanned_pairs.begin(), scanned_pairs.end());
	thrust::device_vector<float> camera_pos_gpu(camera_position, camera_position + 3);
	thrust::device_vector<float> laser_pos_gpu(laser_position, laser_position + 3);
	const uint32_t nvoxels = voxels_per_side[0] * voxels_per_side[1] * voxels_per_side[2];
	thrust::device_vector<float> voxel_volume_gpu(voxel_volume, voxel_volume + nvoxels);
	thrust::device_vector<float> volume_zero_pos_gpu(volume_zero_pos, volume_zero_pos + 3);
	thrust::device_vector<float> voxel_inc_gpu(voxel_inc, voxel_inc + 9);
	thrust::device_vector<float> t0_gpu(&t0, &t0 + 1);
	thrust::device_vector<float> deltaT_gpu(&deltaT, &deltaT + 1);
	thrust::device_vector<uint32_t> voxels_per_side_gpu(voxels_per_side, voxels_per_side + 3);

	{
		hipError_t error = hipGetLastError();
		if (error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			exit(1);
		}
	}

	int blockSize, minGridSize;
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cuda_backprojection_impl, sizeof(double), 0); 
	{
		hipError_t error = hipGetLastError();
		if (error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			exit(1);
		}
	}

	// Limit blocksize to the number of pairs (or else backprojection will fail!)
	blockSize = std::min({(uint32_t) blockSize, num_pairs, 256u});

	// Force a smaller grid size to make each kernel run very short.
	minGridSize = 16;

	std::vector<uint32_t> kernel_voxels(minGridSize * minGridSize * minGridSize * 3);
	for (int x = 0; x < minGridSize; x++)
	for (int y = 0; y < minGridSize; y++)
	for (int z = 0; z < minGridSize; z++)
	{
		kernel_voxels[(x*minGridSize*minGridSize+y*minGridSize+z)*3 + 0] = x;
		kernel_voxels[(x*minGridSize*minGridSize+y*minGridSize+z)*3 + 1] = y;
		kernel_voxels[(x*minGridSize*minGridSize+y*minGridSize+z)*3 + 2] = z;
	}

	uint32_t *kernel_voxels_gpu;
	const uint32_t num_blocks_per_kernel_run = minGridSize*minGridSize*minGridSize;
	hipMalloc((void **)&kernel_voxels_gpu, 3*num_blocks_per_kernel_run*sizeof(uint32_t));
	hipMemcpy(kernel_voxels_gpu, kernel_voxels.data(), 3*num_blocks_per_kernel_run*sizeof(uint32_t), hipMemcpyHostToDevice);

	dim3 xyz_blocks(minGridSize, minGridSize, minGridSize);
	dim3 threads_in_block(blockSize, 1, 1);
	uint32_t number_of_runs = std::ceil(std::max(std::max(voxels_per_side[0], voxels_per_side[1]), voxels_per_side[2]) / (float) minGridSize);
	number_of_runs = number_of_runs * number_of_runs * number_of_runs;

	std::cout << "Backprojecting on the GPU using the \"optimal\" configuration" << std::endl;
	std::cout << "# Blocks: " << xyz_blocks.x << ' ' << xyz_blocks.y << ' ' << xyz_blocks.z << std::endl;
	std::cout << "# Threads per block: " << threads_in_block.x << ' ' << threads_in_block.y << ' ' << threads_in_block.z << std::endl;
	std::cout << "# Kernel calls: " << number_of_runs << std::endl;
	
	auto start = std::chrono::steady_clock::now();
	#if __linux__
	tqdm bar;
	bar.set_theme_braille();
	#else
	std::cout << 0 << " / " << number_of_runs << std::flush;
	#endif
	for (uint32_t r = 0; r < number_of_runs; r++)
	{
		auto start = std::chrono::steady_clock::now();
		cuda_backprojection_impl<<<xyz_blocks, threads_in_block, blockSize*sizeof(double)>>>(
			thrust::raw_pointer_cast(&transient_chunk_gpu[0]),
			thrust::raw_pointer_cast(&T_gpu[0]),
			thrust::raw_pointer_cast(&num_pairs_gpu[0]),
			thrust::raw_pointer_cast(&scanned_pairs_gpu[0]),
			thrust::raw_pointer_cast(&camera_pos_gpu[0]),
			thrust::raw_pointer_cast(&laser_pos_gpu[0]),
			thrust::raw_pointer_cast(&voxel_volume_gpu[0]),
			thrust::raw_pointer_cast(&volume_zero_pos_gpu[0]),
			thrust::raw_pointer_cast(&voxel_inc_gpu[0]),
			thrust::raw_pointer_cast(&t0_gpu[0]),
			thrust::raw_pointer_cast(&deltaT_gpu[0]),
			thrust::raw_pointer_cast(&voxels_per_side_gpu[0]),
			thrust::raw_pointer_cast(&kernel_voxels_gpu[0]));

		hipDeviceSynchronize();
		#if __linux__
		bar.progress(r, number_of_runs);
		#else
		std::cout << '\r' << r+1 << " / " << number_of_runs << std::flush;
		#endif
	}
	hipDeviceSynchronize();
	#if __linux__
	bar.finish();
	#else
	std::cout << std::endl;
	#endif
	auto end = std::chrono::steady_clock::now();
	std::cout << "Backprojection took "
		<< std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count()
		<< " ms" << std::endl;

	// check for errors
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(1);
	}
	
	thrust::copy(voxel_volume_gpu.begin(), voxel_volume_gpu.end(), voxel_volume);
}