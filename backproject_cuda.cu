#include "hip/hip_runtime.h"

#include <chrono>
#include <math.h>
#include <iostream>
#include <thread>
#include <chrono>

#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <hip/hip_complex.h>

#ifdef _WIN32
#define MAKE_DLL_EXPORT __declspec(dllexport)
#endif
#include "backproject_cuda.hpp"
#include "tqdm.h"

__device__
float distance(const float *p1, const float *p2) 
{
	// Checking if either is a nullptr reduces performance dramatically!
	// if (p1 == nullptr || p2 == nullptr) return 0.0;
    float buff[3];
    for (int i = 0; i < 3; i++)
    {
        buff[i] = p1[i] - p2[i];
    }

    return norm3df(buff[0], buff[1], buff[2]);
}

__forceinline__ __device__
void compute_distance(const float * voxel_position,
					   const float *laser_pos, 
					   const float *camera_pos, 
					   const pointpair * pair,
					   float * distance_out)
{
	// From the laser to the wall
	float laser_wall_distance = distance(laser_pos, pair->laser_point);
	// From the wall to the current voxel
	float laser_point_voxel_distance = distance(pair->laser_point, voxel_position);
	// From the wall back to the camera
	float cam_wall_distance = distance(pair->cam_point, camera_pos);
	// From the object back to the wall
	float voxel_cam_point_distance = distance(voxel_position, pair->cam_point);
	
	*distance_out = laser_wall_distance + laser_point_voxel_distance + voxel_cam_point_distance + cam_wall_distance;
}

__forceinline__ __device__
void advance_block(const uint32_t *voxels_per_side, 
				   uint32_t * kernel_voxels, 
				   uint32_t & block_id,
				   uint32_t * xyz,
				   uint32_t & voxel_id)
{
	// First needs to find the x y z coordinates of the voxel
	block_id = (blockIdx.x * gridDim.y * gridDim.z +
				blockIdx.y * gridDim.z +
				blockIdx.z) * 3;
	for (uint32_t i = 0; i < 3; i++)
		xyz[i] = kernel_voxels[block_id+i]; // , kernel_voxels[block_id+1], kernel_voxels[block_id+2]};
	
	// Set the next voxel to be computed by this blockIdx in the next call.
	// We advance on the Z axis by the dimensions of the kernel, overflowing to the Y axis and 
	// then to the X axis. This would match row-major access to the 3D array.
	__syncthreads();
	if (threadIdx.x == 0)
	{
		uint32_t* next_xyz = &kernel_voxels[block_id];
		next_xyz[2] = xyz[2] + gridDim.z;
		if (next_xyz[2] >= voxels_per_side[2])
		{
			next_xyz[2] = next_xyz[2] % voxels_per_side[2];
			next_xyz[1] = next_xyz[1] + gridDim.y;
			if (next_xyz[1] >= voxels_per_side[1])
			{
				next_xyz[1] = next_xyz[1] % voxels_per_side[1];
				next_xyz[0] = next_xyz[0] + gridDim.x;
			}
		}
	}
	__syncthreads();
	voxel_id = xyz[0] * voxels_per_side[1] * voxels_per_side[2] +
			   xyz[1] * voxels_per_side[2] + 
			   xyz[2];
}


__global__
void cuda_backprojection_impl(float *transient_data,
                              uint32_t *T,
                              uint32_t *num_pairs,
                              pointpair *scanned_pairs,
                              float *camera_pos,
                              float *laser_pos,
                              float *voxel_volume,
                              float *volume_zero_pos,
                              float *voxel_inc,
                              float *t0,
                              float *deltaT,
							  uint32_t *voxels_per_side,
							  uint32_t *kernel_voxels)
{
	uint32_t block_id, voxel_id, xyz[3];
	advance_block(voxels_per_side, kernel_voxels, block_id, xyz, voxel_id);

	// If the block is outside the volume don't do anything
	if ((xyz[0] >= voxels_per_side[0]) | (xyz[1] >= voxels_per_side[1]) | (xyz[2] >= voxels_per_side[2]))
		return;
	
	extern __shared__ double local_array[];
	double& radiance_sum = local_array[threadIdx.x];
	radiance_sum = 0.0;

	// Don't run if the current voxel is not 0. This means the current block has already finished.
	if (voxel_volume[voxel_id] == 0.0)
	{
		float voxel_position[] = {volume_zero_pos[0]+voxel_inc[0]*xyz[0],
								  volume_zero_pos[1]+voxel_inc[1]*xyz[1],
								  volume_zero_pos[2]+voxel_inc[2]*xyz[2]};

		for (uint32_t i = 0; i < *num_pairs / blockDim.x; i++)
		{
			uint32_t pair_index = i * blockDim.x + threadIdx.x;
			float total_distance;
			compute_distance(voxel_position, laser_pos, camera_pos, &scanned_pairs[pair_index], &total_distance);
			uint32_t time_index = round((total_distance - *t0) / *deltaT);
			uint32_t tdindex = pair_index * *T + time_index;

			radiance_sum += transient_data[tdindex]; // * distance_attenuation;
		}
	}
    __syncthreads();
	if (threadIdx.x == 0)
	{
		// Compute the reduction in a single thread and write it
		for (int i = 1; i < blockDim.x; i++) {
			local_array[0] += local_array[i];
		}
		voxel_volume[voxel_id] = (float) local_array[0];
	}
    __syncthreads();
}

__global__
void cuda_complex_backprojection_impl(hipComplex *transient_data,
									  uint32_t *T,
									  uint32_t *num_pairs,
									  pointpair *scanned_pairs,
									  float *camera_pos,
									  float *laser_pos,
									  float *voxel_volume,
									  float *volume_zero_pos,
									  float *voxel_inc,
									  float *t0,
									  float *deltaT,
									  uint32_t *voxels_per_side,
									  uint32_t *kernel_voxels)
{
	uint32_t block_id, voxel_id, xyz[3];
	advance_block(voxels_per_side, kernel_voxels, block_id, xyz, voxel_id);

	// If the block is outside the volume don't do anything
	if ((xyz[0] >= voxels_per_side[0]) | (xyz[1] >= voxels_per_side[1]) | (xyz[2] >= voxels_per_side[2]))
		return;
		
	extern __shared__ hipDoubleComplex local_array2[];
	hipDoubleComplex& radiance_sum = local_array2[threadIdx.x];
	radiance_sum = make_hipDoubleComplex(0.0, 0.0);

	// Don't run if the current voxel is not 0. This means the current block has already finished.
	if (voxel_volume[voxel_id] == 0.0)
	{
		float voxel_position[] = {volume_zero_pos[0]+voxel_inc[0]*xyz[0],
								  volume_zero_pos[1]+voxel_inc[1]*xyz[1],
								  volume_zero_pos[2]+voxel_inc[2]*xyz[2]};

		for (uint32_t i = 0; i < *num_pairs / blockDim.x; i++)
		{
			uint32_t pair_index = i * blockDim.x + threadIdx.x;
			float total_distance;
			compute_distance(voxel_position, laser_pos, camera_pos, &scanned_pairs[pair_index], &total_distance);
			uint32_t time_index = round((total_distance - *t0) / *deltaT);
			uint32_t tdindex = pair_index * *T + time_index;

			radiance_sum = hipCadd(radiance_sum, hipComplexFloatToDouble(transient_data[tdindex])); // * distance_attenuation;
		}
	}
    __syncthreads();
	if (threadIdx.x == 0)
	{
		// Compute the reduction in a single thread and write it
		for (int i = 1; i < blockDim.x; i++) {
			local_array2[0] = hipCadd(local_array2[0], local_array2[i]);
		}
		voxel_volume[voxel_id] = hipCreal(local_array2[0]);
	}
    __syncthreads();
}

void call_cuda_backprojection(const float* transient_chunk,
                              uint32_t transient_size, uint32_t T,
                              const std::vector<pointpair> scanned_pairs,
                              const float* camera_position,
                              const float* laser_position,
                              float* voxel_volume,
                              const uint32_t* voxels_per_side,
                              const float* volume_zero_pos,
                              const float* voxel_inc,
                              float t0,
                              float deltaT)
{

	thrust::device_vector<float> transient_chunk_gpu(transient_chunk, transient_chunk + transient_size);
	thrust::device_vector<uint32_t> T_gpu(&T, &T + 1);
	uint32_t num_pairs = scanned_pairs.size();
	thrust::device_vector<uint32_t> num_pairs_gpu(&num_pairs, &num_pairs + 1);
	thrust::device_vector<pointpair> scanned_pairs_gpu(scanned_pairs.begin(), scanned_pairs.end());
	thrust::device_vector<float> camera_pos_gpu(camera_position, camera_position + 3);
	thrust::device_vector<float> laser_pos_gpu(laser_position, laser_position + 3);
	const uint32_t nvoxels = voxels_per_side[0] * voxels_per_side[1] * voxels_per_side[2];
	thrust::device_vector<float> volume_zero_pos_gpu(volume_zero_pos, volume_zero_pos + 3);
	thrust::device_vector<float> voxel_inc_gpu(voxel_inc, voxel_inc + 3);
	thrust::device_vector<float> t0_gpu(&t0, &t0 + 1);
	thrust::device_vector<float> deltaT_gpu(&deltaT, &deltaT + 1);
	thrust::device_vector<uint32_t> voxels_per_side_gpu(voxels_per_side, voxels_per_side + 3);
	
	{
		hipError_t error = hipGetLastError();
		if (error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			exit(1);
		}
	}

	int blockSize, minGridSize;
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cuda_backprojection_impl, sizeof(double), 0); 
	{
		hipError_t error = hipGetLastError();
		if (error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			exit(1);
		}
	}

	// Limit blocksize to the number of pairs (or else backprojection will fail!)
	blockSize = std::min({(uint32_t) blockSize, num_pairs, 256u});

	// Force a smaller grid size to make each kernel run very short.
	minGridSize = 16;

	std::vector<uint32_t> kernel_voxels(minGridSize * minGridSize * minGridSize * 3);
	for (int x = 0; x < minGridSize; x++)
	for (int y = 0; y < minGridSize; y++)
	for (int z = 0; z < minGridSize; z++)
	{
		kernel_voxels[(x*minGridSize*minGridSize+y*minGridSize+z)*3 + 0] = x;
		kernel_voxels[(x*minGridSize*minGridSize+y*minGridSize+z)*3 + 1] = y;
		kernel_voxels[(x*minGridSize*minGridSize+y*minGridSize+z)*3 + 2] = z;
	}

	thrust::device_vector<uint32_t> kernel_voxels_gpu(kernel_voxels.begin(), kernel_voxels.end());

	dim3 xyz_blocks(minGridSize, minGridSize, minGridSize);
	dim3 threads_in_block(blockSize, 1, 1);
	uint32_t number_of_runs = std::ceil(std::max(std::max(voxels_per_side[0], voxels_per_side[1]), voxels_per_side[2]) / (float) minGridSize);
	number_of_runs = number_of_runs * number_of_runs * number_of_runs;

	size_t free_mem, total_mem;
	hipMemGetInfo(&free_mem, &total_mem);
	uint32_t nvoxels_chunk = nvoxels;
	uint32_t nmem_transfers = 1, transfer_period = number_of_runs;
	uint32_t transfers = 0;
	while (nvoxels_chunk*sizeof(float) >= free_mem)
	{
		// TODO: This is GOING TO fail if the volume chunk is smaller than the minGridSize
		nvoxels_chunk = nvoxels_chunk / 2;
		nmem_transfers = nmem_transfers * 2;
		transfer_period = transfer_period / 2;
	}

	thrust::device_vector<float> voxel_volume_gpu(voxel_volume, voxel_volume+nvoxels_chunk);

	std::cout << "Backprojecting on the GPU using the \"optimal\" configuration" << std::endl;
	std::cout << "# Blocks: " << xyz_blocks.x << ' ' << xyz_blocks.y << ' ' << xyz_blocks.z << std::endl;
	std::cout << "# Threads per block: " << threads_in_block.x << ' ' << threads_in_block.y << ' ' << threads_in_block.z << std::endl;
	std::cout << "# Kernel calls: " << number_of_runs << std::endl;
	
	auto start = std::chrono::steady_clock::now();
	tqdm bar;
	bar.set_theme_braille();
	for (uint32_t r = 0; r < number_of_runs; r++)
	{
		if (r > 0 && r % transfer_period == 0)
		{
			thrust::copy(voxel_volume_gpu.begin(), voxel_volume_gpu.end(), voxel_volume+transfers*nvoxels_chunk);
			thrust::copy(voxel_volume+transfers*nvoxels_chunk, voxel_volume+transfers*nvoxels_chunk+nvoxels_chunk, thrust::raw_pointer_cast(&voxel_volume_gpu[0]));
			transfers++;
		}
		auto start = std::chrono::steady_clock::now();
		cuda_backprojection_impl<<<xyz_blocks, threads_in_block, blockSize*sizeof(double)>>>(
			thrust::raw_pointer_cast(&transient_chunk_gpu[0]),
			thrust::raw_pointer_cast(&T_gpu[0]),
			thrust::raw_pointer_cast(&num_pairs_gpu[0]),
			thrust::raw_pointer_cast(&scanned_pairs_gpu[0]),
			thrust::raw_pointer_cast(&camera_pos_gpu[0]),
			thrust::raw_pointer_cast(&laser_pos_gpu[0]),
			thrust::raw_pointer_cast(&voxel_volume_gpu[0]),
			thrust::raw_pointer_cast(&volume_zero_pos_gpu[0]),
			thrust::raw_pointer_cast(&voxel_inc_gpu[0]),
			thrust::raw_pointer_cast(&t0_gpu[0]),
			thrust::raw_pointer_cast(&deltaT_gpu[0]),
			thrust::raw_pointer_cast(&voxels_per_side_gpu[0]),
			thrust::raw_pointer_cast(&kernel_voxels_gpu[0]));
		hipDeviceSynchronize();
		bar.progress(r, number_of_runs);
	}
	hipDeviceSynchronize();
	bar.finish();
	auto end = std::chrono::steady_clock::now();
	std::cout << "Backprojection took "
		<< std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count()
		<< " ms" << std::endl;

	// check for errors
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(1);
	}

	thrust::copy(voxel_volume_gpu.begin(), voxel_volume_gpu.end(), voxel_volume);
}